#include "hip/hip_runtime.h"
#define WARP_SIZE 32
#define USE_DOUBLE
//#define SPECULAR_REFLECTION
//#define USE_RELAXATION
#define GAMMA 267500.0 // ms^-1 * T^-1
#define PI 3.1415926535897932384626433832795
//#define USE_RELAXATION

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <iostream>
#include <iomanip>
#include <vector>
#include <time.h>

#if defined USE_DOUBLE
typedef double real;
#define EPSILON 1e-14
#define GAMMA 267500.0
#define PI 3.1415926535897932384626433832795
#else
typedef float real;
#define EPSILON 1e-6
#define GAMMA 267500.0f
#define PI 3.1415926535897932384626433832795f
#endif

using namespace std;

#include "misc.cuh"
#include "vector3.cuh"
#include "cudaVector.cuh"
#include "timer.cuh"
#include "compare.cuh"
#include "pinnedVector.cuh"
#include "cudaVector.cu"
#include "pinnedVector.cu"
#include "bfunctors.cuh"
#include "substrate.cuh"
#include "cylinderXY.cuh"
#include "Sphere.cuh"
#include "plane.cuh"
#include "empty.cuh"
#include "lattice.cuh"
#include "simuparams.cuh"

#if defined USE_DOUBLE
#include "boundaryCheck.cuh"
#else 
#include "boundaryCheck_float.cuh"
#endif

#include "kernelSetup.cuh"
#include "kernelMag.cuh"
#include "kernelDEBUG.cuh"

#if defined USE_DOUBLE
#include "kernelPhase.cuh"
#else
#include "kernelPhase_float.cuh"
#endif

#include "kernelLattice.cuh"
#include "kernelWC.cuh"
#include "CPUkernels.cuh"
#if defined USE_DOUBLE
#include "gfunctors.cuh"
#else
#include "gfunctors_float.cuh"
#endif

#include "phaseAcquisition.cuh"
#include "phaseAcquisitionStream.cuh"
#include "magAcquisition.cuh"
#include "magAcquisitionStream.cuh"

#if defined USE_DOUBLE
#include "blochdiff.cuh"
#else
#include "blochdiff_float.cuh"
#endif



int main (){

  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "updateWalkersLattice"), hipFuncCachePreferL1 );
  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "updateWalkersPhase"), hipFuncCachePreferL1 );
  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "updateWalkersMag"), hipFuncCachePreferL1 );
  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "setup_kernel"), hipFuncCachePreferL1 );
  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "_functionReduceAtom"), hipFuncCachePreferShared );
  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "_functionTransformAndReduceAtom"), hipFuncCachePreferShared );
  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "_functionTransformAndSumTwoVectorsAtom"), hipFuncCachePreferShared );
	
  int number_of_particles = 32768; //needs to be a factor of two
  real D = 2.5E-6;
  //real D = 1E-6;
  real timestep = .001;  
  real speed = sqrt(6.0*D/timestep);

  //real rad = .005;
  int threads = 128;
  int blocks = number_of_particles/threads;
  
  for (int z = 2; z < 200; z = z + 2){
  int NOI = z;
  int NOM = 2;
  phaseAcquisitionStream<CosGFunc> pas(number_of_particles); 
  
	for (int j = 0; j < NOI; j++){
			
		real gradient_duration= 10;	
		real gradient_spacing  = 2.0;
		real echo_time = 2.0*gradient_duration + gradient_spacing ;
		int number_of_timesteps = (int) (echo_time/timestep);		
		phaseAcquisition<CosGFunc> pa(NOM,number_of_timesteps,number_of_particles,j*time(NULL));
		
		for (int i = 0; i < NOM; i++) {
					
			int N = 1+4*j;
			real G = i*0.0000025*N;
	
			CosGFunc cosGRAD(G, gradient_duration,gradient_spacing, N, Vector3(1.0,0.0,0.0));
			pa.addMeasurement(cosGRAD);	
				
		}
		pas.addAcquisition(pa); 
		
	}
	
  real rad = .0025;
  Sphere spher(0.0,0.0,0.0,rad,0.0,0.0,D,0,0.0);
  
  std::vector<int> plan(3); plan[0] = 0; plan[1] = NOI;  plan[2] = NOI;
  std::vector<int> numOfSMPerDevice(1); numOfSMPerDevice[0] = 14; numOfSMPerDevice[1] = 2; 
 
  CPUtimer timer1, timer2, timer3,timer4;
  timer1.start();
  pas.runAcquisitionStream(spher, timestep, blocks, threads, 1, plan, numOfSMPerDevice); 
  timer1.stop();
  pas.flushADC();

  timer2.start(); 
  for (int i = 0; i < NOI; i++){
	pas.runAcquisitionWC(i,spher, timestep, blocks, threads);
 } 
 timer2.stop(); 
 pas.flushADC();

// timer3.start();
// for (int i = 0; i < NOI; i++){
	// pas.runAcquisitionCPU(i,spher, timestep);
// } 
// timer3.stop();
// pas.flushADC();

timer4.start();
for (int i = 0; i < NOI; i++){
	pas.runAcquisition(i,spher, timestep, blocks,threads,14);
} 
timer4.stop();
pas.flushADC();

std::cout << NOI << " " << timer1.getTime() << " " << timer2.getTime() << " " /* << timer3.getTime() << " " */ << timer4.getTime() <<std::endl;
}


  
  
}
