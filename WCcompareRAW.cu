#define WARP_SIZE 32
#define KAHAN_SUMMATION
//#define USE_DOUBLE
//#define SPECULAR_REFLECTION
//#define USE_RELAXATION
//#define GAMMA 267500.0 // ms^-1 * T^-1
//#define PI 3.1415926535897932384626433832795
//#define USE_RELAXATION

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <iostream>
#include <iomanip>
#include <vector>
#include <time.h>

#if defined USE_DOUBLE
typedef double real;
#define EPSILON 1e-14
#define GAMMA 267500.0
#define PI 3.1415926535897932384626433832795
#else
typedef float real;
#define EPSILON 1e-6
#define GAMMA 267500.0f
#define PI 3.1415926535897932384626433832795f
#endif

using namespace std;

#include "misc.cuh"
#include "vector3.cuh"
#include "cudaVector.cuh"
#include "timer.cuh"
#include "compare.cuh"
#include "pinnedVector.cuh"
#include "cudaVector.cu"
#include "pinnedVector.cu"
#include "bfunctors.cuh"
#include "substrate.cuh"
#include "cylinderXY.cuh"
#include "Sphere.cuh"
#include "plane.cuh"
#include "empty.cuh"
#include "slab.cuh"
#include "lattice.cuh"
#include "simuparams.cuh"

#if defined USE_DOUBLE
#include "boundaryCheck.cuh"
#else 
#include "boundaryCheck_float.cuh"
#endif

#include "kernelSetup.cuh"
#include "kernelMag.cuh"
#include "kernelDEBUG.cuh"

#if defined USE_DOUBLE
#include "kernelPhase.cuh"
#else
#include "kernelPhase_float.cuh"
#endif

#include "kernelLattice.cuh"
#include "kernelWC.cuh"
#include "CPUkernels.cuh"
#if defined USE_DOUBLE
#include "gfunctors.cuh"
#else
#include "gfunctors_float.cuh"
#endif

#include "phaseAcquisition.cuh"
#include "phaseAcquisitionStream.cuh"
#include "magAcquisition.cuh"
#include "magAcquisitionStream.cuh"

#if defined USE_DOUBLE
#include "blochdiff.cuh"
#else
#include "blochdiff_float.cuh"
#endif

int main (){

  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "updateWalkersLattice"), hipFuncCachePreferL1 );
  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "updateWalkersPhase"), hipFuncCachePreferL1 );
  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "updateWalkersMag"), hipFuncCachePreferL1 );
  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "setup_kernel"), hipFuncCachePreferL1 );
  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "_functionReduceAtom"), hipFuncCachePreferShared );
  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "_functionTransformAndReduceAtom"), hipFuncCachePreferShared );
  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "_functionTransformAndSumTwoVectorsAtom"), hipFuncCachePreferShared );
	
  int number_of_particles = 65536; //needs to be a factor of two
  real D = 1.0E-6;
  real timestep = .001;  

  //real rad = .005;
  int threads = 128;
  int blocks = number_of_particles/threads;
  

  phaseAcquisitionStream<SinGFunc> pas(number_of_particles);
  
   int NOI = 100;
   int NOM = 1;

 	real gradient_duration = 10;	
	real gradient_spacing  = 1.0;
	real echo_time = 2.0*gradient_duration + gradient_spacing ;
	int number_of_timesteps = (int) (echo_time/timestep) + 1;		
    phaseAcquisition<SinGFunc> pa(NOM*NOI,number_of_timesteps,number_of_particles,time(NULL));
	real G = .01;
	
	for (int j = 0; j < NOI; j++){
		for (int i = 0; i < NOM; i++) {
			int N = j + 1;
			SinGFunc sinGRAD(G, gradient_duration, gradient_spacing, N, Vector3(1.0,0.0,0.0));
			pa.addMeasurement(sinGRAD);			
		}
	}
	
  pas.addAcquisition(pa);
	
 
  Slab slabby(.005,D);

  for (int i = 0; i < 3; i++){  

  CPUtimer timer1, timer2, timer3;
  timer1.start();
  pas.runAcquisition(0, slabby, timestep, blocks, threads, 14); 
  timer1.stop();
  timer1.display();

  timer2.start(); 
  pas.runAcquisitionWC(0, slabby, timestep, blocks, threads);
  timer2.stop(); 
  timer2.display();
  
  // timer3.start(); 
  // pas.runAcquisitionCPU(0, slabby, timestep);
  // timer3.stop(); 
  // timer3.display();
  
  }
  
  
}
