#include "hip/hip_runtime.h"
#define WARP_SIZE 32
#define USE_DOUBLE
#define SPECULAR_REFLECTION
//#define USE_RELAXATION
#define GAMMA 267500.0 // ms^-1 * T^-1
#define PI 3.1415926535897932384626433832795
//#define USE_RELAXATION

#include <algorithm>
#include <assert.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iomanip>
#include <iostream>
#include <time.h>
#include <vector>

#if defined USE_DOUBLE
typedef double real;
#define EPSILON 1e-14

#else
typedef float real;
#define EPSILON 1e-6

#endif

using namespace std;

#include "CPUkernels.cuh"
#include "RPSinitializer.h"
#include "Sphere.cuh"
#include "bfunctors.cuh"
#include "blochdiff.cuh"
#include "boundaryCheck.cuh"
#include "compare.cuh"
#include "cudaVector.cu"
#include "cudaVector.cuh"
#include "cylinderXY.cuh"
#include "deviates.h"
#include "empty.cuh"
#include "gamma.h"
#include "gfunctors.cuh"
#include "kernelDEBUG.cuh"
#include "kernelLattice.cuh"
#include "kernelMag.cuh"
#include "kernelPhase.cuh"
#include "kernelSetup.cuh"
#include "kernelWC.cuh"
#include "lattice.cuh"
#include "magAcquisition.cuh"
#include "magAcquisitionStream.cuh"
#include "misc.cuh"
#include "nr3.h"
#include "phaseAcquisition.cuh"
#include "phaseAcquisitionStream.cuh"
#include "pinnedVector.cu"
#include "pinnedVector.cuh"
#include "plane.cuh"
#include "ran.h"
#include "simuparams.cuh"
#include "substrate.cuh"
#include "timer.cuh"
#include "vector3.cuh"

int main() {

    hipFuncSetCacheConfig(reinterpret_cast<const void*>("updateWalkersMag"), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>("setup_kernel"), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>("_functionReduceAtom"), hipFuncCachePreferShared);

    int number_of_particles = 57344; // needs to be a factor of two
    real timestep = .001;

    int threads = 128;
    int blocks = number_of_particles / threads;

    phaseAcquisitionStream<CosGFunc> pas(number_of_particles);

    int NOI = 20;
    int NOM = 20;

    real gradient_duration = 20;
    real gradient_spacing = 2.0;
    real echo_time = 2.0 * gradient_duration + gradient_spacing;
    int number_of_timesteps = (int)(echo_time / timestep);
    phaseAcquisition<CosGFunc> pa(NOM * NOI, number_of_timesteps,
                                  number_of_particles, time(NULL));

    for (int j = 0; j < NOI; j++) {
        for (int i = 0; i < NOM; i++) {
            int N = 1 + 10 * j;
            real G = 8. * i * 0.0000025 * N;
            CosGFunc cosGRAD(G, gradient_duration, gradient_spacing, N,
                             Vector3(1.0, 0.0, 0.0));
            pa.addMeasurement(cosGRAD);
        }
    }

    pas.addAcquisition(pa);

    real radius = .006;
    real D_extra = 2.5E-6;
    real D_intra = 1.0E-6;
    real T2_i = 200;
    real T2_e = 200;
    real f = .8;
    real a = sqrt(PI * radius * radius / f);

    std::vector<Cylinder_XY> basis(1);
    Lattice lattice(a, a, a, T2_e, 0.0, D_extra, 1);
    basis[0] = Cylinder_XY(a / 2.0, a / 2.0, radius, T2_i, 0.0, D_intra, 1, 0.0,
                           EPSILON);
    /*
            std::vector<Cylinder_XY> basis;
            Lattice lattice(cube_length, cube_length, cube_length, T2_e, 0.0,
       D_extra,100); RPSLatticeInitializer<Cylinder_XY> rpsli(lattice,0);
            rpsli.gammaRadialDist( 12344124, alpha,  beta, .0001,
       cube_length/10); rpsli.uniformCenterDist( 12344213* 5 );
            rpsli.setRegions();
            rpsli.correctEdges();
            lattice = rpsli.lat; //needed to reinitialize basis size (since it
       was initialized to 100 and there will be > 100 cylinders).

            // if (rpsli.basis.size() != 100){std::cout << " Basis Size Does not
       equal 100 " << std::endl;}
            // std::cout << "lattice basis size = " << lattice.getBasisSize() <<
       std::endl; for (int i = 0; i < rpsli.basis.size(); i++){
                    basis.push_back(Cylinder_XY(0.0, 0.0, 0.0,  T2_i,0.0,
       D_intra, i+1, 0.0)); basis[i].setRadius(rpsli.basis[i].getRadius() );
                    basis[i].setCenter(rpsli.basis[i].getCenter() );
                    basis[i].setEPS( (1E-13));
                    basis[i].setRegion(rpsli.basis[i].getRegion());
                    std::cout << rpsli.basis[i].getCenter()  << "  " <<
       rpsli.basis[i].getRadius() << " " << std::endl;

            }

    */

    std::vector<int> plan(3);
    plan[0] = 0;
    plan[1] = NOI * NOM;
    plan[2] = NOI * NOM;
    std::vector<int> numOfSMPerDevice(1);
    numOfSMPerDevice[0] = 14;
    numOfSMPerDevice[1] = 2;

    pas.runAcquisitionLattice(0, &basis[0], lattice, timestep, blocks, threads,
                              14);

    std::cout << std::endl << " Signals " << std::endl;

    for (int j = 0; j < NOI * NOM; j++) {
        std::cout << setprecision(20);
        std::cout << pas.getAcquisition(0).getGradientFunctors()[j].getFreq()
                  << " ";
        std::cout << pas.getAcquisition(0).getGradientFunctors()[j].getG()
                  << " ";
        std::cout << pas.getAcquisition(0).getMx()[j] << " "
                  << pas.getAcquisition(0).getMy()[j] << " ";
        std::cout << std::endl;
    }
}
