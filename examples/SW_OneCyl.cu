#include "hip/hip_runtime.h"
#define WARP_SIZE 32
#define USE_DOUBLE
#define SPECULAR_REFLECTION
//#define USE_RELAXATION
#define GAMMA 267500.0 // ms^-1 * T^-1
#define PI 3.1415926535897932384626433832795
//#define USE_RELAXATION

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <iostream>
#include <iomanip>
#include <vector>
#include <time.h>
#include <algorithm>

#if defined USE_DOUBLE
typedef double real;
#define EPSILON 1e-14

#else
typedef float real;
#define EPSILON 1e-6

#endif

using namespace std;

#include "misc.cuh"
#include "vector3.cuh"
#include "cudaVector.cuh"
#include "timer.cuh"
#include "compare.cuh"
#include "pinnedVector.cuh"
#include "cudaVector.cu"
#include "pinnedVector.cu"
#include "bfunctors.cuh"
#include "substrate.cuh"
#include "cylinderXY.cuh"
#include "Sphere.cuh"
#include "plane.cuh"
#include "empty.cuh"
#include "lattice.cuh"
#include "simuparams.cuh"
#include "boundaryCheck.cuh"
#include "kernelSetup.cuh"
#include "kernelMag.cuh"
#include "kernelDEBUG.cuh"
#include "kernelPhase.cuh"
#include "kernelLattice.cuh"
#include "kernelWC.cuh"
#include "CPUkernels.cuh"
#include "gfunctors.cuh"
#include "phaseAcquisition.cuh"
#include "phaseAcquisitionStream.cuh"
#include "magAcquisition.cuh"
#include "magAcquisitionStream.cuh"
#include "blochdiff.cuh"
#include "nr3.h"
#include "ran.h"
#include "gamma.h"
#include "deviates.h"
#include "RPSinitializer.h"



int main (){

  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "updateWalkersMag"), hipFuncCachePreferL1 );
  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "setup_kernel"), hipFuncCachePreferL1 );
  hipFuncSetCacheConfig(reinterpret_cast<const void*>( "_functionReduceAtom"), hipFuncCachePreferShared );

  int number_of_particles = 57344; //needs to be a factor of two
  real timestep = .001;  

  int threads = 128;
  int blocks = number_of_particles/threads;
	
  phaseAcquisitionStream<SWOGSEFunc> pas(number_of_particles);
  phaseAcquisitionStream<SWOGSEFunc> pas1; 
  phaseAcquisitionStream<SWOGSEFunc> pas2;
  phaseAcquisitionStream<SWOGSEFunc> pas3;
  phaseAcquisitionStream<SWOGSEFunc> pas4;
  phaseAcquisitionStream<SWOGSEFunc> pas5;
  phaseAcquisitionStream<SWOGSEFunc> pas6;
  phaseAcquisitionStream<SWOGSEFunc> pas7;
  phaseAcquisitionStream<SWOGSEFunc> pas8;
  phaseAcquisitionStream<SWOGSEFunc> pas9;
  phaseAcquisitionStream<SWOGSEFunc> pas10;
  
   int NOI = 20;
   int NOM = 20;

 	real gradient_duration = 30;	
	real gradient_spacing  = 2.0;
	real echo_time = 2.0*gradient_duration + gradient_spacing ;
	int number_of_timesteps = (int) (echo_time/timestep);		
    phaseAcquisition<SWOGSEFunc> pa(NOM*NOI,number_of_timesteps,number_of_particles,time(NULL));
	
	for (int j = 0; j < NOI; j++){
		for (int i = 0; i < NOM; i++) {
			int N = 1+j;
			real G = i*0.0000025*N;
			SWOGSEFunc cosGRAD(G, gradient_duration,gradient_spacing, N, Vector3(1.0,0.0,0.0));
			pa.addMeasurement(cosGRAD);			
		}
	}
	
	pas.addAcquisition(pa);

	real radius = .0015;
	real D_extra = 2.5E-6;
	real D_intra = 1.0E-6;
	real T2_i = 200;
	real T2_e = 200;
	real f = .6;
	real a = sqrt( PI*radius*radius / f );

	
	std::vector<Cylinder_XY> basis(1);
	Lattice lattice(a, a, a, T2_e, 0.0, D_extra,1);
	basis[0] = Cylinder_XY(a/2.0, a/2.0,  radius,  T2_i,0.0, D_intra, 1, 0.0, EPSILON);
/*	
	std::vector<Cylinder_XY> basis;
	Lattice lattice(cube_length, cube_length, cube_length, T2_e, 0.0, D_extra,100);
	RPSLatticeInitializer<Cylinder_XY> rpsli(lattice,0);
	rpsli.gammaRadialDist( 12344124, alpha,  beta, .0001, cube_length/10);
	rpsli.uniformCenterDist( 12344213* 5 );
	rpsli.setRegions();
	rpsli.correctEdges();
	lattice = rpsli.lat; //needed to reinitialize basis size (since it was initialized to 100 and there will be > 100 cylinders).
	
	// if (rpsli.basis.size() != 100){std::cout << " Basis Size Does not equal 100 " << std::endl;}
	// std::cout << "lattice basis size = " << lattice.getBasisSize() << std::endl;
	for (int i = 0; i < rpsli.basis.size(); i++){
		basis.push_back(Cylinder_XY(0.0, 0.0, 0.0,  T2_i,0.0, D_intra, i+1, 0.0));
		basis[i].setRadius(rpsli.basis[i].getRadius() );
		basis[i].setCenter(rpsli.basis[i].getCenter() );
		basis[i].setEPS( (1E-13));
		basis[i].setRegion(rpsli.basis[i].getRegion());
		std::cout << rpsli.basis[i].getCenter()  << "  " << rpsli.basis[i].getRadius() << " " << std::endl;
		
	}	
	
*/

  std::vector<int> plan(3); plan[0] = 0; plan[1] = NOI;  plan[2] = NOI;
  std::vector<int> numOfSMPerDevice(1); numOfSMPerDevice[0] = 14; numOfSMPerDevice[1] = 2; 


  pas1 = pas; pas1.getAcquisition(0).getSeed() *= 2;
   pas2 = pas; pas2.getAcquisition(0).getSeed() *= 3;
   pas3 = pas; pas3.getAcquisition(0).getSeed() *= 4;
   pas4 = pas; pas4.getAcquisition(0).getSeed() *= 5;
   pas5 = pas; pas4.getAcquisition(0).getSeed() *= 6;
   pas6 = pas; pas4.getAcquisition(0).getSeed() *= 7;
   pas7 = pas; pas4.getAcquisition(0).getSeed() *= 8;
   pas8 = pas; pas4.getAcquisition(0).getSeed() *= 9;
   pas9 = pas; pas4.getAcquisition(0).getSeed() *= 10;
  
   pas.runAcquisitionLattice(0, &basis[0], lattice,  timestep, blocks, threads, 14);
   pas1.runAcquisitionLattice(0, &basis[0], lattice,  timestep, blocks, threads, 14);
   pas2.runAcquisitionLattice(0, &basis[0], lattice,  timestep, blocks, threads, 14);
   pas3.runAcquisitionLattice(0, &basis[0], lattice,  timestep, blocks, threads, 14);
   pas4.runAcquisitionLattice(0, &basis[0], lattice,  timestep, blocks, threads, 14);
   pas5.runAcquisitionLattice(0, &basis[0], lattice,  timestep, blocks, threads, 14);
   pas6.runAcquisitionLattice(0, &basis[0], lattice,  timestep, blocks, threads, 14);
   pas7.runAcquisitionLattice(0, &basis[0], lattice,  timestep, blocks, threads, 14);
   pas8.runAcquisitionLattice(0, &basis[0], lattice,  timestep, blocks, threads, 14);
   pas9.runAcquisitionLattice(0, &basis[0], lattice,  timestep, blocks, threads, 14);

  std::cout << std::endl << " Signals " << std::endl;
 
  for (int j = 0; j < NOI*NOM; j++){
	std::cout << setprecision(20);
	std::cout << pas.getAcquisition(0).getGradientFunctors()[j].getFreq() << " " ;
	std::cout << pas.getAcquisition(0).getGradientFunctors()[j].getG() << " " ;
	std::cout << pas.getAcquisition(0).getMx()[j] << " " << pas.getAcquisition(0).getMy()[j] << " " ;
	std::cout << pas1.getAcquisition(0).getMx()[j] << " " << pas1.getAcquisition(0).getMy()[j] << " " ;
	std::cout << pas2.getAcquisition(0).getMx()[j] << " " << pas2.getAcquisition(0).getMy()[j] << " " ;
	std::cout << pas3.getAcquisition(0).getMx()[j] << " " << pas3.getAcquisition(0).getMy()[j] << " " ;
	std::cout << pas4.getAcquisition(0).getMx()[j] << " " << pas4.getAcquisition(0).getMy()[j] << " " ;
	std::cout << pas5.getAcquisition(0).getMx()[j] << " " << pas5.getAcquisition(0).getMy()[j] << " " ;
	std::cout << pas6.getAcquisition(0).getMx()[j] << " " << pas6.getAcquisition(0).getMy()[j] << " " ;
	std::cout << pas7.getAcquisition(0).getMx()[j] << " " << pas7.getAcquisition(0).getMy()[j] << " " ;
	std::cout << pas8.getAcquisition(0).getMx()[j] << " " << pas8.getAcquisition(0).getMy()[j] << " " ;
	std::cout << pas9.getAcquisition(0).getMx()[j] << " " << pas9.getAcquisition(0).getMy()[j] << " " ;
	std::cout << std::endl;
  }


  
 
}
