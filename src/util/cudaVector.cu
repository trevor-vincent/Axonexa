#include "hip/hip_runtime.h"
#include "reduction.cuh"

// log all errors into logger

template <class T> cudaVector<T>::cudaVector() {
    alloc = false;
    copied = false;
    device = 0;
}

template <class T> cudaVector<T>::cudaVector(int size) {
    _v.resize(size);
    safe_cuda(hipMalloc((void **)&dev_ptr, size * sizeof(T)));
    alloc = true;
    copied = false;
    device = 0;
}

template <class T> cudaVector<T>::cudaVector(int size, T val) {
    _v = std::vector<T>(size, val);
    safe_cuda(hipMalloc((void **)&dev_ptr, size * sizeof(T)));
    alloc = true;
    copied = false;
    device = 0;
}

template <class T> cudaVector<T>::~cudaVector() {

    if (alloc) {
        safe_cuda(hipSetDevice(device));
        safe_cuda(hipFree(dev_ptr));
    }
}

template <class T> void cudaVector<T>::setDevice(int _device) {
    device = _device;
}

template <class T> void cudaVector<T>::copyToDevice() {
    if (alloc == false) {
        safe_cuda(hipMalloc((void **)&dev_ptr, _v.size() * sizeof(T)));
        alloc = true;
    }
    safe_cuda(hipMemcpy(dev_ptr, &_v[0], _v.size() * sizeof(T),
                         hipMemcpyHostToDevice));
    copied = true;
}

template <class T> void cudaVector<T>::copyFromDevice() {
    if (alloc == true) {
        safe_cuda(hipMemcpy(&_v[0], dev_ptr, _v.size() * sizeof(T),
                             hipMemcpyDeviceToHost));
    }
}

template <class T> void cudaVector<T>::malloc(int asize) {
    safe_cuda(hipMalloc((void **)&dev_ptr, asize * sizeof(T)));
    alloc = true;
    _v.resize(asize);
}

template <class T> int cudaVector<T>::size() { return _v.size(); }

template <class T> void cudaVector<T>::resize(int size) { _v.resize(size); }

template <class T> T &cudaVector<T>::operator[](int i) { return _v[i]; }

template <class T> void cudaVector<T>::operator=(pinnedVector<T> &v) {

    int iter;
    if (v.size() >= _v.size()) {
        iter = _v.size();
    } else {
        iter = v.size();
    }

    for (int i = 0; i < iter; i++) {
        _v[i] = v[i];
    }
}
template <class T> T *cudaVector<T>::getPointer() { return dev_ptr; }

template <class T> void cudaVector<T>::copyTo(std::vector<T> &v) {

    int iter;
    if (v.size() >= _v.size()) {
        iter = _v.size();
    } else {
        iter = v.size();
    }

    for (int i = 0; i < iter; i++) {
        v[i] = _v[i];
    }
}

template <class T> void cudaVector<T>::operator=(std::vector<T> &v) {

    int iter;
    if (v.size() >= _v.size()) {
        iter = _v.size();
    } else {
        iter = v.size();
    }

    for (int i = 0; i < iter; i++) {
        _v[i] = v[i];
    }
}

template <class T>
void cudaVector<T>::sum(cudaScalar<T> &sum, int threads, int blocks,
                        hipStream_t stream) {
    _functionReduceAtom<<<blocks, threads, sizeof(T) * (threads - WARP_SIZE),
                          stream>>>(
        sum.getPointer(), cudaVector<T>::getPointer(), cudaVector<T>::size());
}

// sum ( X )
template <class T>
void cudaVector<T>::sum(cudaVector<T> &sum, int threads, int blocks, int chunks,
                        int sizeOfChunk, hipStream_t stream) {
    for (int i = 0; i < chunks; i++) {
        _functionReduceAtom<<<blocks, threads,
                              sizeof(T) * (threads - WARP_SIZE), stream>>>(
            sum.getPointer() + i, cudaVector<T>::getPointer() + i * sizeOfChunk,
            sizeOfChunk);
    }
}

// sum ( transform ( X ) )
template <class T>
template <class Transform>
void cudaVector<T>::transformAndSum(cudaScalar<T> &sum, int threads, int blocks,
                                    hipStream_t stream) {
    _functionTransformAndReduceAtom<T, Transform>
        <<<blocks, threads, sizeof(T) * (threads - WARP_SIZE), stream>>>(
            sum.getPointer(), cudaVector<T>::getPointer(),
            cudaVector<T>::size());
}

// sum ( transform ( X ) )
template <class T>
template <class Transform>
void cudaVector<T>::transformAndSum(cudaVector<T> &sum, int threads, int blocks,
                                    int chunks, int sizeOfChunk,
                                    hipStream_t stream) {
    for (int i = 0; i < chunks; i++) {
        _functionTransformAndReduceAtom<T, Transform>
            <<<blocks, threads, sizeof(T) * (threads - WARP_SIZE), stream>>>(
                sum.getPointer() + i,
                cudaVector<T>::getPointer() + i * sizeOfChunk, sizeOfChunk);
    }
}

// sum ( transform ( X , Y ) )
template <class T>
template <class Transform>
void cudaVector<T>::transformAndSumTwoVectors(cudaVector<T> &sum,
                                              cudaVector<T> &secondVector,
                                              int threads, int blocks,
                                              int chunks, int sizeOfChunk,
                                              hipStream_t stream) {
    for (int i = 0; i < chunks; i++) {
        _functionTransformAndSumTwoVectorsAtom<T, Transform>
            <<<blocks, threads, sizeof(T) * (threads - WARP_SIZE), stream>>>(
                sum.getPointer() + i,
                cudaVector<T>::getPointer() + i * sizeOfChunk,
                secondVector.getPointer(), sizeOfChunk);
    }
}
